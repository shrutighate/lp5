#include "hip/hip_runtime.h"
%%cu
#include <iostream>
using namespace std;


// CUDA code to multiply matrices
__global__ void multiply(int* A, int* B, int* C, int size) {
    // Uses thread indices and block indices to compute each element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int i = 0; i < size; i++) {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}


void initialize(int* matrix, int size) {
    for (int i = 0; i < size * size; i++) {
        matrix[i] = rand() % 10;
    }
}


void print(int* matrix, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            cout << matrix[row * size + col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';
}


int main() {
    int* A, * B, * C;

    int N = 2;
    int blockSize =  16;

    int matrixSize = N * N;
    size_t matrixBytes = matrixSize * sizeof(int);

    A = new int[matrixSize];
    B = new int[matrixSize];
    C = new int[matrixSize];

    initialize(A, N);
    initialize(B, N);
    cout << "Matrix A: \n";
    print(A, N);

    cout << "Matrix B: \n";
    print(B, N);

    
    int* X, * Y, * Z;
    // Allocate space
    hipMalloc(&X, matrixBytes);
    hipMalloc(&Y, matrixBytes);
    hipMalloc(&Z, matrixBytes);

    // Copy values from A to X
    hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);
    
    // Copy values from A to X and B to Y
    hipMemcpy(Y, B, matrixBytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 2;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    int BLOCKS = N / THREADS;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // Launch kernel
    multiply<<<blocks, threads>>>(X, Y, Z, N);

    hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);
    cout << "Multiplication of matrix A and B: \n";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}



// This CUDA C++ code demonstrates matrix multiplication using GPU parallelism. Let's break down the code:

// 1. **Kernel Function (`multiply`)**:
//    - This kernel function is responsible for computing the product of two matrices `A` and `B`.
//    - It takes pointers to the input matrices `A` and `B`, as well as the output matrix `C`, and the size of the matrices (`size`).
//    - Each thread computes one element of the output matrix `C`.
//    - The thread indices (`row` and `col`) are computed using block and thread indices, and each thread iterates over the corresponding row of matrix `A` and column of matrix `B` to calculate the dot product.

// 2. **Helper Functions (`initialize` and `print`)**:
//    - `initialize`: This function initializes a matrix with random values between 0 and 9. It takes a pointer to the matrix and its size as arguments.
//    - `print`: This function prints the elements of a matrix. It takes a pointer to the matrix and its size as arguments.

// 3. **Main Function**:
//    - The main function initializes matrices `A` and `B`, prints them, and then performs matrix multiplication using CUDA.
//    - Matrices `A` and `B` are initialized with random values using the `initialize` function and printed using the `print` function.
//    - Device memory (`X`, `Y`, and `Z`) is allocated using `hipMalloc` for matrices `A`, `B`, and `C` respectively.
//    - The values of matrices `A` and `B` are copied from host to device memory using `hipMemcpy`.
//    - The number of threads per block (`THREADS`) is set to 2, and the number of blocks per grid (`BLOCKS`) is calculated based on the size of the matrices.
//    - The kernel function `multiply` is launched with the specified number of blocks and threads per block.
//    - The result matrix `C` is copied back from device to host memory using `hipMemcpy` and printed.
//    - Finally, memory allocated on the device is freed using `hipFree`.

// 4. **Output**:
//    - The code prints the matrices `A` and `B` before multiplication, and the result matrix `C` after multiplication.

// 5. **Memory Management**:
//    - Memory allocated on the device (`X`, `Y`, and `Z`) is freed at the end of the main function to release GPU resources.

// Overall, this code demonstrates how to leverage GPU parallelism using CUDA to perform matrix multiplication, which can significantly accelerate computation for large matrices compared to sequential CPU-based computation.




































%%cu
#include <iostream>
using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}


void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 4;
    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, vectorSize);
    initialize(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}



// This CUDA C++ code performs vector addition using GPU parallelism. Let's go through the code step by step:

// 1. **Kernel Function (`add`)**:
//    - The kernel function is responsible for adding corresponding elements of two input vectors `A` and `B` and storing the result in vector `C`.
//    - Each thread is assigned a unique thread ID (`tid`) calculated based on the block index and thread index.
//    - Each thread performs the addition operation for one element of the vectors, checking first if the thread ID is within the bounds of the vectors.

// 2. **Helper Functions (`initialize` and `print`)**:
//    - `initialize`: This function initializes a vector with random values between 0 and 9. It takes a pointer to the vector and its size as arguments.
//    - `print`: This function prints the elements of a vector. It takes a pointer to the vector and its size as arguments.

// 3. **Main Function**:
//    - The main function initializes vectors `A` and `B`, prints them, and then performs vector addition using CUDA.
//    - Vectors `A` and `B` are initialized with random values using the `initialize` function and printed using the `print` function.
//    - Device memory (`X`, `Y`, and `Z`) is allocated using `hipMalloc` for vectors `A`, `B`, and `C` respectively.
//    - The values of vectors `A` and `B` are copied from host to device memory using `hipMemcpy`.
//    - The number of threads per block (`threadsPerBlock`) is set to 256, and the number of blocks per grid (`blocksPerGrid`) is calculated based on the size of the vectors.
//    - The kernel function `add` is launched with the specified number of blocks and threads per block.
//    - The result vector `C` is copied back from device to host memory using `hipMemcpy` and printed.
//    - Finally, memory allocated on the device is freed using `hipFree`.

// 4. **Output**:
//    - The code prints vectors `A` and `B` before addition, and the result vector `C` after addition.

// 5. **Memory Management**:
//    - Memory allocated on the device (`X`, `Y`, and `Z`) is freed at the end of the main function to release GPU resources.

// This code demonstrates how to leverage GPU parallelism using CUDA to perform vector addition, which can significantly accelerate computation for large vectors compared to sequential CPU-based computation.
